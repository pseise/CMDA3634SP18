#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"




  __device__ unsigned int CUDAModProd(unsigned int a, unsigned int b, unsigned int p){
    unsigned int za = a;
    unsigned int ab = 0;
  
    while (b > 0) {
      if (b%2 == 1) ab = (ab +  za) % p;
      za = (2 * za) % p;
      b /= 2;
    }
    return ab;
  }

  __device__ unsigned int CUDAModExp(unsigned int a, unsigned int b, unsigned int p){

    unsigned int z = a;
    unsigned int aExpb = 1;
  
    while (b > 0) {
      if (b%2 == 1) aExpb = CUDAModProd(aExpb, z, p);
      z = CUDAModProd(z, z, p);
      b /= 2;
    }
    return aExpb;

  }


  __global__  void zaxbys(int p,int g, int h, unsigned int *secretKey)
  {
      for (unsigned int n = blockIdx.x * blockDim.x + threadIdx.x; n <p-1; n = n + blockDim.x * gridDim.x) {

        if(CUDAModExp(g, n+1, p) == h){

          printf("Secret Key found! x = %u \n", n+1);
          
            secretKey[0] = n+1;

        }
  }
  }
     

  //declare storage for an ElGamal cryptosytem
  
  
int main (int argc, char **argv) {
   unsigned int n, p, g, h, x;
   unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  
    FILE *filly = fopen("public_key.txt", "r");
    fscanf(filly, "%u\n%u\n%u\n%u", &n,&p,&g,&h);

    FILE *messy = fopen("message.txt", "r");
    fscanf(messy, "%u\n", &Nints);
    
   
    
    unsigned int cpi = n/8;
    
    unsigned int *ary1 = (unsigned int *)malloc(Nints*sizeof(unsigned int));
    
    unsigned int *ary2 = (unsigned int *)malloc(Nints*sizeof(unsigned int));

    
    
    
    

  for(int x = 0; x<Nints; x++){

    fscanf(messy, "%u %u\n", &ary1[x], &ary2[x]);
  }

  
  // find the secret key
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    int Nthreads = 32;

    int Nblocks = (p+Nthreads-1)/Nthreads;

    unsigned int *h_x = (unsigned int *) malloc(Nints*sizeof(unsigned int));

    unsigned int *d_a;

    hipMalloc(&d_a, sizeof(unsigned int));
    double startTime = clock();
    zaxbys <<<Nblocks,Nthreads>>> (p,g,h,d_a);
    hipDeviceSynchronize();
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;



    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
    
    hipMemcpy(h_x, d_a, sizeof(unsigned int), hipMemcpyDeviceToHost);

    x = *h_x;
  }

  
  ElGamalDecrypt(ary1, ary2, Nints, p, x);

  unsigned int Numchrs = cpi * Nints;

  unsigned char *origins = (unsigned char *) malloc(Numchrs*sizeof(unsigned char));

  convertZToString(ary1, Nints, origins, Numchrs);

  printf("The message is %s\n", origins);

  fclose(messy);
  fclose(filly);
  



  



  return 0;
}
